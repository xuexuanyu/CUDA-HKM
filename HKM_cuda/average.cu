#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" 
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "average.h"
#include <stdio.h>
#include "ccCommon.hpp"
#include "ccDistance.hpp"


__global__ void getblockid_Kernel(uint* arr_div, uint* ids, uint size, uint* nid_div)
{
	int i = threadIdx.x;
	uint m=i*(size/5);
	uint sum=0;
	int j;
	for (j = 0; j < size; ++j)
	{
		if (ids[j] == i)
		{
			arr_div[m] = j;
			sum++;
			m++;

		}
	}
	nid_div[i] = sum;

}

__global__ void getmean_Kernel(uint *arr_div, uint *data, uint *mean_div, uint *nid_div)
{
	int k = blockIdx.x;
	int dim = threadIdx.x;
	int size;
	size=nid_div[k];
	uint tmean=0;
	uint m = 0;
	for (int i = 0; i < k-1; ++i)
	{
		m += nid_div[i-1];
	}
	for(int i = 0; i < size; ++i)
	{
		uint id = arr_div[m+i];
		tmean += data[id * 64 + dim];
	}
	mean_div[k*64+dim] = tmean;
	
}

template <typename T>
hipError_t getidWithCuda(Data<T>& data1, uint* ids, uint* nid_host, uint *d_host)
{

	hipError_t cudaStatus;
	uint nids = data1.size();
	uint **addr_div;
	uint *nid_div;
	uint *ids_div;
	uint *arr_div;
	cudaStatus = hipMalloc((void**)&arr_div, nids * 2 * sizeof(uint));
	cudaStatus = hipMalloc((void**)&ids_div, nids*sizeof(uint));
	cudaStatus = hipMalloc((void**)&nid_div, sizeof(uint) * 10);
	cudaStatus = hipMemcpy(ids_div, ids, nids*sizeof(uint), hipMemcpyHostToDevice);
	getblockid_Kernel << <1, 10 >> >(arr_div,ids_div, nids, nid_div);
	cudaStatus = hipMemcpy(nid_host, nid_div, sizeof(uint)*10, hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(d_host, arr_div, nids * 2 * sizeof(uint), hipMemcpyDeviceToHost);
	int sum=0;
	for (int i = 0; i < 10; ++i)
	{
		sum += nid_host[i];
	}
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy failed!");
	}

	hipFree(arr_div);
	hipFree(ids_div);
	hipFree(nid_div);
	//remalloc
	return cudaStatus;
}




template <typename T>
void getmeanbycuda(Data<T>& data1, uint* ids, uint *meansbycuda)
{
	hipError_t cudaStatus;
	//cudaStatus = hipDeviceReset();

	uint *nid_host=new uint[10];
	uint size = data1.size();
	uint *ids_arr = new uint[size*2];
	cudaStatus=getidWithCuda(data1, ids, nid_host, ids_arr);//get ids_arr
	//-----re-arr--------------
	uint *idarray = new uint[size];
	uint m = 0;
	for (int i = 0; i < 10; i++)
	{
		for (int j = 0; j < nid_host[i]; ++j)
		{
			idarray[m] = ids_arr[i*(size / 5) + j];
			m++;
		}
	}
	//------get need data-----------------
	uint nbytes = data1.size() * data1.ndims;
	uint *data_host = new uint[nbytes];
	for (int i = 0; i < data1.size(); ++i)
	{
		uint pid = data1.filterId(i);
		for (int j = 0; j < data1.ndims; ++j)
		{
			data_host[i*data1.ndims + j] = data1.data.fixed.data.full[pid * data1.ndims + j];
		}

	}
	//-----------------
	uint *data2;
	uint *mean_div;
	uint *nid_div;
	uint *arr_div;
	uint *mean_host = new uint[64 * 10];
	cudaStatus = hipMalloc((void**)&nid_div, sizeof(uint) * 10);
	cudaStatus = hipMalloc((void**)&data2, nbytes*sizeof(uint));
	cudaStatus = hipMalloc((void**)&mean_div, 10 * 64 * sizeof(uint));
	cudaStatus = hipMalloc((void**)&arr_div, size*sizeof(uint));
	cudaStatus = hipMemcpy(arr_div, idarray, size*sizeof(uint), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(data2, data_host, nbytes*sizeof(uint), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(nid_div, nid_host, sizeof(uint) * 10, hipMemcpyHostToDevice);
	
	getmean_Kernel << <10, 64 >> >(arr_div, data2, mean_div, nid_div);
	//cudaStatus = hipMemcpy(mean_host, mean_div, 10 * 64 * sizeof(uint), hipMemcpyDeviceToHost);
	/*for (int i = 0; i < 10; ++i)
	{
		for (int j = 0; j < 64; ++j)
		{
			meansbycuda[i * 64 + j] = (mean_host[i * 64 + j]) / nid_host[i];
		}
	}*/
	
	
	delete[] data_host;
	delete[] ids_arr;
	delete[] idarray;
	delete[] nid_host;
	
}
/*
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, size_t size)
{
	int *dev_a = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// Launch a kernel on the GPU with one thread for each element.
		addKernel << <1, size, size * sizeof(int), 0 >> >(dev_c, dev_a);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	return cudaStatus;
}*/


#define GETMEAN_F(T)      \
  template void getmeanbycuda(Data<T>& ,uint*,uint *);
#define GETMEAN_D(T)      \
  template void getmeanbycuda(Data<T>& ,uint*,uint *);

TEMPLATE(GETMEAN_F)
TEMPLATE(GETMEAN_D)